#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "rt_types.hpp"

// Kernel to convert from conservative to primitive variables
__global__ void primFromConsKernel(float *d_r, float *d_ru, float *d_rv, float *d_e,
    float *d_u, float *d_v, float *d_p, float *d_c,
    SimParams params) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Modified: Process both interior AND ghost cells
    // Remove this line: if (i >= params.Nx || j >= params.Ny) return;

    // Process all cells including ghosts (-1 to Nx and -1 to Ny)
    if (i <= params.Nx && j <= params.Ny && i >= -1 && j >= -1) {
    int idx_ij = idx(i, j, params.Nx, params.Ny);

    // Get density
    float r = d_r[idx_ij];

    // Ensure density is positive
    r = fmaxf(r, 1e-6f);

    // Calculate velocities
    float u = d_ru[idx_ij] / r;
    float v = d_rv[idx_ij] / r;

    // Store velocities
    d_u[idx_ij] = u;
    d_v[idx_ij] = v;

    // Calculate pressure
    float kinetic_energy = 0.5f * r * (u*u + v*v);
    float p = (params.gamma - 1.0f) * (d_e[idx_ij] - kinetic_energy);

    // Ensure pressure is positive
    p = fmaxf(p, 1e-6f);

    // Store pressure
    d_p[idx_ij] = p;

    // Calculate sound speed
    d_c[idx_ij] = sqrtf(params.gamma * p / r);
    }
}

void computePrimitiveVariables(float *d_r, float *d_ru, float *d_rv, float *d_e,
    float *d_u, float *d_v, float *d_p, float *d_c,
    SimParams params) {
    // Modified: Include ghost cells in calculation
    dim3 blockDim(16, 16);
    dim3 gridDim((params.Nx + 2 + blockDim.x - 1) / blockDim.x,
    (params.Ny + 2 + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    primFromConsKernel<<<gridDim, blockDim>>>(d_r, d_ru, d_rv, d_e, d_u, d_v, d_p, d_c, params);

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
    fprintf(stderr, "Error in computePrimitiveVariables: %s\n", hipGetErrorString(err));
    }
}